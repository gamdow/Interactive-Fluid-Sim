#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include "hip/hip_vector_types.h"
#include "hip/hip_runtime_api.h"

#include <iostream> // for host code
#include <stdio.h> // for kernel code

float const PI = 3.14159265359f;

template<class T>
TextureObject<T>::TextureObject()
  : __buffer(nullptr)
  , __pitch(0u)
  , __object(0u)
{}

// Initialise the Texture Object required by advect's interpolated sampling.
template<class T>
void TextureObject<T>::init(int3 _buffer_spec) {
  checkCudaErrors(hipMallocPitch(&__buffer, &__pitch, sizeof(T) * _buffer_spec.x, _buffer_spec.y));
  hipResourceDesc resDesc; memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = __buffer;
  resDesc.res.pitch2D.pitchInBytes = __pitch;
  resDesc.res.pitch2D.width = _buffer_spec.x;
  resDesc.res.pitch2D.height = _buffer_spec.y;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<T>();
  hipTextureDesc texDesc; memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  checkCudaErrors(hipCreateTextureObject(&__object, &resDesc, &texDesc, nullptr));

  std::cout << "f1Object " << __buffer << " " << __pitch << " " << __object << std::endl;
}

template<class T>
void TextureObject<T>::shutdown() {
  checkCudaErrors(hipDestroyTextureObject(__object));
  checkCudaErrors(hipFree(__buffer));
}

__global__ void advect_velocity(float2 * o_velocity, hipTextureObject_t _velocityObj, int3 _buffer_spec, float _dt, float2 _rdx);
__global__ void calc_divergence(float * o_divergence, float2 const * _velocity, float const * _fluid, int3 _buffer_spec, float2 _r2dx);
__global__ void pressure_decay(float * io_pressure, float const * _fluid, int3 _buffer_spec);
__global__ void pressure_solve(float * o_pressure, float const * _pressure, float const * _divergence, float const * _fluid, int3 _buffer_spec, float2 _dx);
__global__ void sub_gradient(float2 * io_velocity, float const * _pressure, float const * _fluid, int3 _buffer_spec, float2 _r2dx);
__global__ void enforce_slip(float2 * io_velocity, float const * _fluid, int3 _buffer_spec);
__global__ void hsv_to_rgba(hipSurfaceObject_t o_surface, float2 const * _array, float _power, int3 _buffer_spec);
__global__ void d_to_rgba(hipSurfaceObject_t o_surface, float const * _array, float _multiplier, int3 _buffer_spec);

Kernels::Kernels(int2 _dims, int _buffer) {
  std::cout << std::endl;
  reportCapability();
  std::cout << std::endl;
  optimiseBlockSize(_dims, _buffer);
  std::cout << std::endl;
  __f1Object.init(__buffer_spec);
  __f2Object.init(__buffer_spec);
}

Kernels::~Kernels() {
  __f1Object.shutdown();
  __f2Object.shutdown();
}

void Kernels::reportCapability() const {
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  for(int dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    std::cout << "CUDA Device: " << dev << ":" << deviceProp.name << std::endl;
    std::cout << "\tCapability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    int driverVersion = 0, runtimeVersion = 0;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    std::cout << "\tRuntime/Driver: " << runtimeVersion << "/" << driverVersion << std::endl;
  }
}

// Use CUDA's occupancy to determine the optimal blocksize and adjust the screen (and therefore array) resolution to be an integer multiple (then there's no need for bounds checking in the kernels).
void Kernels::optimiseBlockSize(int2 _dims, int _buffer) {
  std::cout << "Desired Resolution: " << _dims.x << " x " << _dims.y << std::endl;
  int N = _dims.x * _dims.y;
  int blockSize, minGridSize;   hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pressure_solve, 0, N);
  __block = dim3(32u, blockSize / 32u);
  std::cout << "Optimal Block: " << __block.x << " x " << __block.y << std::endl;
  __grid = dim3(_dims.x / __block.x, _dims.y / __block.y);
  __dims = make_int2(__grid.x * __block.x, __grid.y * __block.y);
  std::cout << "Adjusted Resolution: " << __dims.x << " x " << __dims.y << std::endl;
  __buffer_spec = make_int3(__dims.x + 2 * _buffer, __dims.y + 2 * _buffer, _buffer);
  __buffered_size = __buffer_spec.x * __buffer_spec.y;
}

struct Index {
  __device__ Index(int3 _buffer_spec)
    : x(blockIdx.x * blockDim.x + threadIdx.x + _buffer_spec.z)
    , y(blockIdx.y * blockDim.y + threadIdx.y + _buffer_spec.z)
    , idx(_buffer_spec.x * y + x)
  {}
  int x, y, idx;
};

struct Stencil :public Index {
  __device__ Stencil(int3 _buffer_spec)
    : Index(_buffer_spec)
    , stencil(idx + make_int4(1, -1, _buffer_spec.x, -_buffer_spec.x))
  {}
  int4 stencil;
};

__global__ void advect_velocity(float2 * o_velocity, hipTextureObject_t _velocityObj, int3 _buffer_spec, float _dt, float2 _rdx) {
  Index ih(_buffer_spec);
  float s = (float)ih.x + 0.5f;
  float t = (float)ih.y + 0.5f;
  float2 pos = make_float2(s, t) - _dt * _rdx * tex2D<float2>(_velocityObj, s, t);
  o_velocity[ih.idx] = tex2D<float2>(_velocityObj, pos.x, pos.y);
}

__global__ void apply_advection(float * o_data, hipTextureObject_t _dataObj, float2 const * _velocity, int3 _buffer_spec, float _dt, float2 _rdx) {
  Index ih(_buffer_spec);
  float2 pos = make_float2(ih.x + 0.5f, ih.y + 0.5f) - _dt * _rdx * _velocity[ih.idx];
  o_data[ih.idx] = tex2D<float>(_dataObj, pos.x, pos.y);
}

__global__ void calc_divergence(float * o_divergence, float2 const * _velocity, float const * _fluid, int3 _buffer_spec, float2 _r2dx) {
  Stencil ih(_buffer_spec);
  o_divergence[ih.idx] = (_velocity[ih.stencil.x].x * _fluid[ih.stencil.x] - _velocity[ih.stencil.y].x * _fluid[ih.stencil.y]) * _r2dx.x
    + (_velocity[ih.stencil.z].y * _fluid[ih.stencil.z] - _velocity[ih.stencil.w].y * _fluid[ih.stencil.w]) * _r2dx.y;
}

__global__ void pressure_decay(float * io_pressure, float const * _fluid, int3 _buffer_spec) {
  Index ih(_buffer_spec);
  io_pressure[ih.idx] *= _fluid[ih.idx] * 0.1f + 0.9f;
}

__global__ void pressure_solve(float * o_pressure, float const * _pressure, float const * _divergence, float const * _fluid, int3 _buffer_spec, float2 _dx) {
  Stencil ih(_buffer_spec);
  o_pressure[ih.idx] = (1.0f / 4.0f) * (
    (4.0f - _fluid[ih.stencil.x] - _fluid[ih.stencil.y] - _fluid[ih.stencil.z] - _fluid[ih.stencil.w]) * _pressure[ih.idx]
    + _fluid[ih.stencil.x] * _pressure[ih.stencil.x]
    + _fluid[ih.stencil.y] * _pressure[ih.stencil.y]
    + _fluid[ih.stencil.z] * _pressure[ih.stencil.z]
    + _fluid[ih.stencil.w] * _pressure[ih.stencil.w]
    - _divergence[ih.idx] * _dx.x * _dx.y);
}

__global__ void sub_gradient(float2 * io_velocity, float const * _pressure, float const * _fluid, int3 _buffer_spec, float2 _r2dx) {
  Stencil ih(_buffer_spec);
  io_velocity[ih.idx] -= _fluid[ih.idx] * _r2dx * make_float2( _pressure[ih.stencil.x] - _pressure[ih.stencil.y], _pressure[ih.stencil.z] - _pressure[ih.stencil.w]);
}

__global__ void enforce_slip(float2 * io_velocity, float const * _fluid, int3 _buffer_spec) {
  Stencil ih(_buffer_spec);
  if(_fluid[ih.idx] > 0.0f) {
    float xvel = _fluid[ih.stencil.x] * _fluid[ih.stencil.y] == 0.0f
      ? ((1.f - _fluid[ih.stencil.x]) * io_velocity[ih.stencil.x].x + (1.f - _fluid[ih.stencil.y]) * io_velocity[ih.stencil.y].x) / (2.f - _fluid[ih.stencil.x] - _fluid[ih.stencil.y])
      : io_velocity[ih.idx].x;
    float yvel = _fluid[ih.stencil.z] * _fluid[ih.stencil.w] == 0.0f
      ? ((1.f - _fluid[ih.stencil.z]) * io_velocity[ih.stencil.z].y + (1.f - _fluid[ih.stencil.w]) * io_velocity[ih.stencil.w].y) / (2.f - _fluid[ih.stencil.z] - _fluid[ih.stencil.w])
      : io_velocity[ih.idx].y;
    io_velocity[ih.idx] = make_float2(xvel, yvel);
  } else {
    io_velocity[ih.idx] = make_float2(0.0f, 0.0f);
  }
}

__global__ void hsv_to_rgba(hipSurfaceObject_t o_surface, float2 const * _array, float _power, int3 _buffer_spec) {
  Index ih(_buffer_spec);
  float h = 6.0f * (atan2f(-_array[ih.idx].x, -_array[ih.idx].y) / (2 * PI) + 0.5);
  float v = __powf(_array[ih.idx].x * _array[ih.idx].x + _array[ih.idx].y * _array[ih.idx].y, _power);
  float hi = floorf(h);
  float f = h - hi;
  float q = v * (1 - f);
  float t = v * f;
  float4 rgb = {.0f, .0f, .0f, 1.0f};
  if(hi == 0.0f || hi == 6.0f) {
    rgb.x = v;
    rgb.y = t;
	} else if(hi == 1.0f) {
    rgb.x = q;
    rgb.y = v;
	} else if(hi == 2.0f) {
		rgb.y = v;
    rgb.z = t;
	} else if(hi == 3.0f) {
		rgb.y = q;
    rgb.z = v;
	} else if(hi == 4.0f) {
    rgb.x = t;
    rgb.z = v;
	} else {
    rgb.x = v;
    rgb.z = q;
  }
  surf2Dwrite(rgb, o_surface, (ih.x - _buffer_spec.z) * sizeof(float4), (ih.y - _buffer_spec.z));
}

__global__ void d_to_rgba(hipSurfaceObject_t o_surface, float const * _array, float _multiplier, int3 _buffer_spec) {
  Index ih(_buffer_spec);
  float pos = (_array[ih.idx] + abs(_array[ih.idx])) / 2.0f;
  float neg = -(_array[ih.idx] - abs(_array[ih.idx])) / 2.0f;
  float4 rgb = make_float4(neg * _multiplier, pos * _multiplier, 0.0, 1.0f);
  surf2Dwrite(rgb, o_surface, (ih.x - _buffer_spec.z) * sizeof(float4), (ih.y - _buffer_spec.z));
}

__global__ void sum_arrays(float2 * o_array, float _c1, float2 const * _array1, float _c2, float2 const * _array2, int3 _buffer_spec) {
  Index ih(_buffer_spec);
  o_array[ih.idx] = _c1 * _array1[ih.idx] + _c2 * _array2[ih.idx];
}

void Kernels::advectVelocity(float2 * io_velocity, float2 _rdx, float _dt) {
  hipMemcpy2D(__f2Object.__buffer, __f2Object.__pitch, io_velocity, sizeof(float2) * __buffer_spec.x, sizeof(float2) * __buffer_spec.x, __buffer_spec.y, hipMemcpyDeviceToDevice);
  advect_velocity<<<__grid,__block>>>(io_velocity, __f2Object.__object, __buffer_spec, _dt, _rdx);
}

void Kernels::advectVelocity(float2 * o_velocity, float2 const * _velocity, float2 _rdx, float _dt) {
  hipMemcpy2D(__f2Object.__buffer, __f2Object.__pitch, _velocity, sizeof(float2) * __buffer_spec.x, sizeof(float2) * __buffer_spec.x, __buffer_spec.y, hipMemcpyDeviceToDevice);
  advect_velocity<<<__grid,__block>>>(o_velocity, __f2Object.__object, __buffer_spec, _dt, _rdx);
}

void Kernels::applyAdvection(float * io_data, float2 const * _velocity, float _dt, float2 _rdx) {
  hipMemcpy2D(__f1Object.__buffer, __f1Object.__pitch, io_data, sizeof(float) * __buffer_spec.x, sizeof(float) * __buffer_spec.x, __buffer_spec.y, hipMemcpyDeviceToDevice);
  apply_advection<<<__grid,__block>>>(io_data, __f1Object.__object, _velocity, __buffer_spec, _dt, _rdx);
}

void Kernels::calcDivergence(float * o_divergence, float2 const * _velocity, float const * _fluid, float2 _r2dx) {
  calc_divergence<<<__grid,__block>>>(o_divergence, _velocity, _fluid, __buffer_spec, _r2dx);
}

void Kernels::pressureDecay(float * io_pressure, float const * _fluid) {
  pressure_decay<<<__grid,__block>>>(io_pressure, _fluid, __buffer_spec);
}

void Kernels::pressureSolve(float * o_pressure, float const * _pressure, float const * _divergence, float const * _fluid, float2 _dx) {
  pressure_solve<<<__grid,__block>>>(o_pressure, _pressure, _divergence, _fluid, __buffer_spec, _dx);
}

void Kernels::subGradient(float2 * io_velocity, float const * _pressure, float const * _fluid, float2 _r2dx) {
  sub_gradient<<<__grid,__block>>>(io_velocity, _pressure, _fluid, __buffer_spec, _r2dx);
}

void Kernels::enforceSlip(float2 * io_velocity, float const * _fluid) {
  enforce_slip<<<__grid,__block>>>(io_velocity, _fluid, __buffer_spec);
}

void Kernels::hsv2rgba(hipSurfaceObject_t o_surface, float2 const * _array, float _power) {
  hsv_to_rgba<<<__grid,__block>>>(o_surface, _array, _power, __buffer_spec);
}

void Kernels::v2rgba(hipSurfaceObject_t o_surface, float const * _array, float _multiplier) {
  d_to_rgba<<<__grid,__block>>>(o_surface, _array, _multiplier, __buffer_spec);
}

// Ax = b
__global__ void jacobi_solve(float * _b, float * _validCells, int3 _buffer_spec, float alpha, float beta, float * _x, float * o_x) {
  Stencil ih(_buffer_spec);
  float xL = _validCells[ih.stencil.y] > 0 ? _x[ih.stencil.y] : _x[ih.idx];
  float xR = _validCells[ih.stencil.x] > 0 ? _x[ih.stencil.x] : _x[ih.idx];
  float xB = _validCells[ih.stencil.w] > 0 ? _x[ih.stencil.w] : _x[ih.idx];
  float xT = _validCells[ih.stencil.z] > 0 ? _x[ih.stencil.z] : _x[ih.idx];
  o_x[ih.idx] = beta * (xL + xR + xB + xT + alpha * _b[ih.idx]);
}

void Kernels::sum(float2 * o_array, float _c1, float2 const * _array1, float _c2, float2 const * _array2) {
  sum_arrays<<<__grid,__block>>>(o_array, _c1, _array1, _c2, _array2, __buffer_spec);
}
