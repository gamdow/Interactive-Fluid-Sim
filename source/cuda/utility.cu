#include "hip/hip_runtime.h"
#include "utility.cuh"

#include "../kernels/kernels.cuh"

void reportCudaCapability() {
  std::cout << "CUDA Capability: " << std::endl;
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  for(int dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int driverVersion; hipDriverGetVersion(&driverVersion);
    int runtimeVersion; hipRuntimeGetVersion(&runtimeVersion);
    std::cout << "\tDevice: " << dev << ": " << deviceProp.name << std::endl;
    std::cout << "\t\tCapability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    std::cout << "\t\tRuntime/Driver: " << runtimeVersion << "/" << driverVersion << std::endl;
  }
}

OptimalBlockConfig::OptimalBlockConfig(Resolution _res)
{
  std::cout << "Optimising Blocksize:" << std::endl;
  // Use CUDA's occupancy to determine the optimal blocksize and adjust the screen (and therefore array) resolution to be an integer multiple (then there's no need for bounds checking in the kernels).
  int blockSize, minGridSize; hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pressure_solve, 0, _res.size);
  block = dim3(32u, blockSize / 32u);
  Resolution(block.x, block.y).print("\tOptimal Block");
  grid = dim3(_res.width / block.x, _res.height / block.y);
  optimal_res = Resolution(grid.x * block.x, grid.y * block.y);
  _res.print("\tDesired Resolution");
  optimal_res.print("\tAdjusted Resolution");
}

void print(std::ostream & _out, float4 _v) {
  _out << "(" << _v.x << "," << _v.y << "," << _v.z << "," << _v.w << ")";
}
