#include "hip/hip_runtime.h"
#include "utility.h"

#include "../debug.h"

__global__ void pressure_solve(float * o_pressure, float const * _pressure, float const * _divergence, float const * _fluid, Resolution _buffer_res, float2 _dx);

void reportCudaCapability() {
  format_out << "CUDA Capability: " << std::endl;
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  for(int dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int driverVersion; hipDriverGetVersion(&driverVersion);
    int runtimeVersion; hipRuntimeGetVersion(&runtimeVersion);
    OutputIndent indent1;
    format_out << "Device: " << dev << ": " << deviceProp.name << std::endl;
    OutputIndent indent2;
    format_out << "Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    format_out << "Runtime/Driver: " << runtimeVersion << "/" << driverVersion << std::endl;
  }
}

OptimalBlockConfig::OptimalBlockConfig(Resolution _res)
{
  format_out << "Optimising Blocksize:" << std::endl;
  // Use CUDA's occupancy to determine the optimal blocksize and adjust the screen (and therefore array) resolution to be an integer multiple (then there's no need for bounds checking in the kernels).
  int blockSize, minGridSize; hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pressure_solve, 0, _res.size);
  block = dim3(32u, blockSize / 32u);
  OutputIndent indent1;
  Resolution(block.x, block.y).print("Optimal Block");
  grid = dim3(_res.width / block.x, _res.height / block.y);
  optimal_res = Resolution(grid.x * block.x, grid.y * block.y);
  _res.print("Desired Resolution");
  optimal_res.print("Adjusted Resolution");
}

__global__ void copy_to_surface(hipSurfaceObject_t o_surface, Resolution _surface_res, uchar3 const * _buffer, Resolution _buffer_res) {
  // surf2Dwrite<uchar3>(_buffer[_buffer_res.idx()], o_surface, (int)(_buffer_res.x() * sizeof(unsigned char)), _buffer_res.y(), hipBoundaryModeTrap);
  #ifdef __CUDA_ARCH__
    __nv_tex_surf_handler("__surf2Dwrite_v2", (typename __nv_surf_trait<uchar3>::cast_type)&_buffer[_buffer_res.idx()], (int)sizeof(uchar3), o_surface, (int)(_buffer_res.x() * sizeof(uchar3)), _buffer_res.y(),  hipBoundaryModeTrap);
  #endif /* __CUDA_ARCH__ */
}

void copyToSurface(OptimalBlockConfig const & _block_config, hipSurfaceObject_t o_surface, Resolution const & _surface_res, uchar3 const * _buffer, Resolution const & _buffer_res) {
  copy_to_surface<<<_block_config.grid, _block_config.block>>>(o_surface, _surface_res, _buffer, _buffer_res);
}

__global__ void copy_to_surface(hipSurfaceObject_t o_surface, Resolution _surface_res, float const * _buffer, Resolution _buffer_res) {
  surf2Dwrite<float>(_buffer[_buffer_res.idx()], o_surface, (_buffer_res.x()) * sizeof(float), _buffer_res.y());
}

void copyToSurface(OptimalBlockConfig const & _block_config, hipSurfaceObject_t o_surface, Resolution const & _surface_res, float const * _buffer, Resolution const & _buffer_res) {
  copy_to_surface<<<_block_config.grid, _block_config.block>>>(o_surface, _surface_res, _buffer, _buffer_res);
}

__global__ void copy_to_surface(hipSurfaceObject_t o_surface, Resolution _surface_res, unsigned char const * _buffer, Resolution _buffer_res) {
  surf2Dwrite<unsigned char>(_buffer[_buffer_res.idx()], o_surface, (_buffer_res.x()) * sizeof(unsigned char), _buffer_res.y());
}

void copyToSurface(OptimalBlockConfig const & _block_config, hipSurfaceObject_t o_surface, Resolution const & _surface_res, unsigned char const * _buffer, Resolution const & _buffer_res) {
  copy_to_surface<<<_block_config.grid, _block_config.block>>>(o_surface, _surface_res, _buffer, _buffer_res);
}

__global__ void copy_to_surface2(hipSurfaceObject_t o_surface, Resolution _surface_res, float4 const * _buffer, Resolution _buffer_res) {
  surf2Dwrite<float4>(_buffer[_buffer_res.idx()], o_surface, (_buffer_res.x()) * sizeof(float4), _buffer_res.y());
}

void copyToSurface(OptimalBlockConfig const & _block_config, hipSurfaceObject_t o_surface, Resolution const & _surface_res, float4 const * _buffer, Resolution const & _buffer_res) {
  copy_to_surface2<<<_block_config.grid, _block_config.block>>>(o_surface, _surface_res, _buffer, _buffer_res);
}

void print(std::ostream & _out, float4 _v) {
  _out << "(" << _v.x << "," << _v.y << "," << _v.z << "," << _v.w << ")";
}
