#include "utility.cuh"

#include "../kernels/kernels.cuh"

void reportCudaCapability() {
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  for(int dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int driverVersion; hipDriverGetVersion(&driverVersion);
    int runtimeVersion; hipRuntimeGetVersion(&runtimeVersion);
    std::cout << "CUDA Device: " << dev << ": " << deviceProp.name << std::endl;
    std::cout << "\tCapability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    std::cout << "\tRuntime/Driver: " << runtimeVersion << "/" << driverVersion << std::endl;
  }
}

OptimalBlockConfig::OptimalBlockConfig(Resolution _res)
{
  std::cout << "Optimising Blocksize:" << std::endl;
  // Use CUDA's occupancy to determine the optimal blocksize and adjust the screen (and therefore array) resolution to be an integer multiple (then there's no need for bounds checking in the kernels).
  _res.print("\tDesired Resolution");
  int blockSize, minGridSize; hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pressure_solve, 0, _res.size);
  block = dim3(32u, blockSize / 32u);
  Resolution(block.x, block.y).print("\tOptimal Block");
  grid = dim3(_res.width / block.x, _res.height / block.y);
  optimal_res = Resolution(grid.x * block.x, grid.y * block.y);
  optimal_res.print("\tAdjusted Resolution");
}
