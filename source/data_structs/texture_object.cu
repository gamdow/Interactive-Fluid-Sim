#include "hip/hip_runtime.h"
#include "texture_object.cuh"

#include <iostream>
#include <typeinfo>
#include <hip/hip_runtime.h>

#include "../cuda/hip/hip_runtime_api.h"

template<class T>
TextureObject<T>::TextureObject()
  : __buffer(nullptr)
  , __pitch(0u)
  , __object(0u)
{}

template<class T>
void TextureObject<T>::init(Resolution const & _res) {
  std::cout << "\tcudaMallocPitch(" << typeid(T).name() << "): ";
  checkCudaErrors(hipMallocPitch(&__buffer, &__pitch, sizeof(T) * _res.width, _res.height));
  std::cout << _res.height * __pitch << " bytes (" << __buffer << ")" << std::endl;
  hipResourceDesc resDesc; memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = __buffer;
  resDesc.res.pitch2D.pitchInBytes = __pitch;
  resDesc.res.pitch2D.width = _res.width;
  resDesc.res.pitch2D.height = _res.height;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<T>();
  hipTextureDesc texDesc; memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  checkCudaErrors(hipCreateTextureObject(&__object, &resDesc, &texDesc, nullptr));
}

template<class T>
void TextureObject<T>::shutdown() {
  checkCudaErrors(hipDestroyTextureObject(__object));
  checkCudaErrors(hipFree(__buffer));
}

#define EXPLICT_INSTATIATION(TYPED_MACRO) \
  TYPED_MACRO(float) \
  TYPED_MACRO(float2) \
  TYPED_MACRO(float3) \
  TYPED_MACRO(float4)

#define TEMPLATE(T) template class TextureObject<T>;
  EXPLICT_INSTATIATION(TEMPLATE)
#undef TEMPLATE
