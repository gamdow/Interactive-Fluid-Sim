#include "hip/hip_runtime.h"
#include "kernels.cuh"

// #include <iostream> // for host code
#include <stdio.h> // for kernel code

#include "../cuda/hip/hip_vector_types.h"
#include "../cuda/hip/hip_runtime_api.h"
#include "../cuda/utility.cuh"

float const PI = 3.14159265359f;

template<typename T>
__device__
inline T lerp(T a, T b, T l) {
  //return (1. - l) * a + l * b;
  return fma(l, b, fma(-l, a, a));
}

__global__ void advect_velocity(float2 * o_velocity, hipTextureObject_t _velocityObj, Resolution _buffer_res, float _dt, float2 _rdx) {
  float s = (float)_buffer_res.x() + 0.5f;
  float t = (float)_buffer_res.y() + 0.5f;
  float2 pos = make_float2(s, t) - _dt * _rdx * tex2D<float2>(_velocityObj, s, t);
  o_velocity[_buffer_res.idx()] = tex2D<float2>(_velocityObj, pos.x, pos.y);
}

template<typename T>
__global__ void apply_advection(T * o_data, hipTextureObject_t _dataObj, float2 const * _velocity, float const * _fluid, Resolution _buffer_res, float _dt, float2 _rdx) {
  if(_fluid[_buffer_res.idx()] > 0.0f) {
    float2 pos = make_float2(_buffer_res.x() + 0.5f, _buffer_res.y() + 0.5f) - _dt * _rdx * _velocity[_buffer_res.idx()];
    o_data[_buffer_res.idx()] = tex2D<T>(_dataObj, pos.x, pos.y);
  } else {
    o_data[_buffer_res.idx()] *= 0.9f;
  }
}

#define TEMPLATE(T) template __global__ void apply_advection(T * o_data, hipTextureObject_t _dataObj, float2 const * _velocity, float const * _fluid, Resolution _buffer_res, float _dt, float2 _rdx);
TEMPLATE(float)
TEMPLATE(float2)
TEMPLATE(float4)
#undef TEMPLATE

__global__ void calc_divergence(float * o_divergence, float2 const * _velocity, float const * _fluid, Resolution _buffer_res, float2 _rdx) {
  int4 const stencil = _buffer_res.stencil();
  o_divergence[_buffer_res.idx()] = (_velocity[stencil.x].x - _velocity[stencil.y].x) * (_rdx.x / 2.0f) + (_velocity[stencil.z].y - _velocity[stencil.w].y) * (_rdx.y / 2.0f);
}

__global__ void pressure_decay(float * io_pressure, float const * _fluid, Resolution _buffer_res) {
  int const idx = _buffer_res.idx();
  io_pressure[idx] *= _fluid[idx] * 0.1f + 0.9f;
}

__global__ void pressure_solve(float * o_pressure, float const * _pressure, float const * _divergence, float const * _fluid, Resolution _buffer_res, float2 _dx) {
  int const idx = _buffer_res.idx();
  int4 const stencil = _buffer_res.stencil();
  float pR = lerp(_pressure[idx], _pressure[stencil.x], _fluid[stencil.x]);
  float pL = lerp(_pressure[idx], _pressure[stencil.y], _fluid[stencil.y]);
  float pU = lerp(_pressure[idx], _pressure[stencil.z], _fluid[stencil.z]);
  float pD = lerp(_pressure[idx], _pressure[stencil.w], _fluid[stencil.w]);
  o_pressure[idx] = (1.0f / 4.0f) * (pR + pL + pU + pD
    - _divergence[idx] * _dx.x * _dx.y);
}

__global__ void sub_gradient(float2 * io_velocity, float const * _pressure, float const * _fluid, Resolution _buffer_res, float2 _rdx) {
  int const idx = _buffer_res.idx();
  int4 const stencil = _buffer_res.stencil();
  float pR = lerp(_pressure[idx], _pressure[stencil.x], _fluid[stencil.x]);
  float pL = lerp(_pressure[idx], _pressure[stencil.y], _fluid[stencil.y]);
  float pU = lerp(_pressure[idx], _pressure[stencil.z], _fluid[stencil.z]);
  float pD = lerp(_pressure[idx], _pressure[stencil.w], _fluid[stencil.w]);
  io_velocity[idx] -= _fluid[idx] * (_rdx / 2.0f) * make_float2(pR - pL, pU - pD);
}

__global__ void enforce_slip(float2 * io_velocity, float const * _fluid, Resolution _buffer_res) {
  int const idx = _buffer_res.idx();
  int4 const stencil = _buffer_res.stencil();
  if(_fluid[idx] > 0.0f) {
    float xvel = _fluid[stencil.x] * _fluid[stencil.y] == 0.0f
      ? ((1.f - _fluid[stencil.x]) * io_velocity[stencil.x].x + (1.f - _fluid[stencil.y]) * io_velocity[stencil.y].x) / (2.f - _fluid[stencil.x] - _fluid[stencil.y])
      : io_velocity[idx].x;
    float yvel = _fluid[stencil.z] * _fluid[stencil.w] == 0.0f
      ? ((1.f - _fluid[stencil.z]) * io_velocity[stencil.z].y + (1.f - _fluid[stencil.w]) * io_velocity[stencil.w].y) / (2.f - _fluid[stencil.z] - _fluid[stencil.w])
      : io_velocity[idx].y;
    io_velocity[idx] = make_float2(xvel, yvel);
  } else {
    io_velocity[idx] = make_float2(0.0f, 0.0f);
  }
}

__global__ void d_to_rgba(float4 * o_buffer, float const * _buffer, Resolution _buffer_res, float _multiplier) {
  int const idx = _buffer_res.idx();
  float pos = (_buffer[idx] + abs(_buffer[idx])) / 2.0f;
  float neg = -(_buffer[idx] - abs(_buffer[idx])) / 2.0f;
  o_buffer[idx] = make_float4(neg * _multiplier, pos * _multiplier, 0.0, 1.0f);
}

// __global__ void d_to_rgba(hipSurfaceObject_t o_surface, Resolution _surface_res, float const * _buffer, Resolution _buffer_res, float _multiplier) {
//   int const idx = _buffer_res.idx();
//   float pos = (_buffer[idx] + abs(_buffer[idx])) / 2.0f;
//   float neg = -(_buffer[idx] - abs(_buffer[idx])) / 2.0f;
//   float4 rgb = make_float4(neg * _multiplier, pos * _multiplier, 0.0, 1.0f);
//   rgb.w = fmin(rgb.x + rgb.y + rgb.z, 1.f);
//   int buffer_diff = _surface_res.buffer - _buffer_res.buffer;
//   surf2Dwrite(rgb, o_surface, (_buffer_res.x() + buffer_diff) * sizeof(float4), _buffer_res.y() + buffer_diff);
// }

// Render 2D field (i.e. velocity) by treating as HSV (hue=direction, saturation=1, value=magnitude) and converting to RGBA
__global__ void hsv_to_rgba(float4 * o_buffer, float2 const * _buffer, Resolution _buffer_res, float _power) {
  int const idx = _buffer_res.idx();
  float h = 6.0f * (atan2f(-_buffer[idx].x, -_buffer[idx].y) / (2 * PI) + 0.5);
  float v = __powf(_buffer[idx].x * _buffer[idx].x + _buffer[idx].y * _buffer[idx].y, _power);
  float hi = floorf(h);
  float f = h - hi;
  float q = v * (1 - f);
  float t = v * f;
  float4 rgb = {.0f, .0f, .0f, 1.0f};
  if(hi == 0.0f || hi == 6.0f) {
    rgb.x = v;
    rgb.y = t;
	} else if(hi == 1.0f) {
    rgb.x = q;
    rgb.y = v;
	} else if(hi == 2.0f) {
		rgb.y = v;
    rgb.z = t;
	} else if(hi == 3.0f) {
		rgb.y = q;
    rgb.z = v;
	} else if(hi == 4.0f) {
    rgb.x = t;
    rgb.z = v;
	} else {
    rgb.x = v;
    rgb.z = q;
  }
  rgb.w = fmin(rgb.x + rgb.y + rgb.z, 1.f);
  o_buffer[idx] = rgb;
  // int buffer_diff = _surface_res.buffer - _buffer_res.buffer;
  // surf2Dwrite(rgb, o_surface, (_buffer_res.x() + buffer_diff) * sizeof(float4), _buffer_res.y() + buffer_diff);
}

// Render 4D field by operating on it with a 4x3 matrix, where the rows are RGB values (a colour for each dimension).
__global__ void float4_to_rgba(float4 * o_buffer, float4 const * _buffer, Resolution _buffer_res, float3 const * _map) {
  int const idx = _buffer_res.idx();
  float4 rgb = make_float4(
    _buffer[idx].x * _map[0].x + _buffer[idx].y * _map[1].x + _buffer[idx].z * _map[2].x + _buffer[idx].w * _map[3].x,
    _buffer[idx].x * _map[0].y + _buffer[idx].y * _map[1].y + _buffer[idx].z * _map[2].y + _buffer[idx].w * _map[3].y,
    _buffer[idx].x * _map[0].z + _buffer[idx].y * _map[1].z + _buffer[idx].z * _map[2].z + _buffer[idx].w * _map[3].z,
    0.75f * (_buffer[idx].x + _buffer[idx].y + _buffer[idx].z + _buffer[idx].w)
  );
  rgb.w = fmin(rgb.x + rgb.y + rgb.z, 1.f);
  o_buffer[idx] = rgb;
  // int buffer_diff = _surface_res.buffer - _buffer_res.buffer;
  // surf2Dwrite(rgb, o_surface, (_buffer_res.x() + buffer_diff) * sizeof(float4), _buffer_res.y() + buffer_diff);
}

__global__ void copy_to_surface(hipSurfaceObject_t o_surface, Resolution _surface_res, float4 const * _buffer, Resolution _buffer_res) {
  int buffer_diff = _surface_res.buffer - _buffer_res.buffer;
  surf2Dwrite(_buffer[_buffer_res.idx()], o_surface, (_buffer_res.x() + buffer_diff) * sizeof(float4), _buffer_res.y() + buffer_diff);
}

__global__ void sum_arrays(float2 * o_array, float _c1, float2 const * _array1, float _c2, float2 const * _array2, Resolution _buffer_res) {
  int const idx = _buffer_res.idx();
  o_array[idx] = _c1 * _array1[idx] + _c2 * _array2[idx];
}

__device__ inline float minmod(float a, float b) {
  return a * b > 0
    ? (a > 0
      ? (a < b ? a : b)
      : (a > b ? a : b))
    : 0;
}

__device__ inline float2 minmod2(float2 a, float2 b) {
  return make_float2(minmod(a.x, b.x), minmod(a.y, b.y));
}

__device__ inline float2 limit_select(float2 * _e1, float2 * _e2, int i, int j) {
  return make_float2(_e2[j].x * _e2[j].x > _e1[j].x * _e1[j].x ? _e1[j].x : _e1[i].x, _e2[j].y * _e2[j].y > _e1[j].y * _e1[j].y ? _e1[j].y : _e1[i].y);
}

__global__ void limit_advection(float2 * o_e, float2 * _e1, float2 * _e2, Resolution _buffer_res) {
  int4 const stencil = _buffer_res.stencil();
  o_e[_buffer_res.idx()] = minmod2(
    minmod2(limit_select(_e1, _e2, _buffer_res.idx(), stencil.x), limit_select(_e1, _e2, _buffer_res.idx(), stencil.y)),
    minmod2(limit_select(_e1, _e2, _buffer_res.idx(), stencil.z), limit_select(_e1, _e2, _buffer_res.idx(), stencil.w)));
}

// // Ax = b
// __global__ void jacobi_solve(float * _b, float * _validCells, Resolution _buffer_res, float alpha, float beta, float * _x, float * o_x) {
//   int const idx = _buffer_res.idx();
//   int4 const stencil = _buffer_res.stencil();
//   float xL = _validCells[stencil.y] > 0 ? _x[stencil.y] : _x[idx];
//   float xR = _validCells[stencil.x] > 0 ? _x[stencil.x] : _x[idx];
//   float xB = _validCells[stencil.w] > 0 ? _x[stencil.w] : _x[idx];
//   float xT = _validCells[stencil.z] > 0 ? _x[stencil.z] : _x[idx];
//   o_x[idx] = beta * (xL + xR + xB + xT + alpha * _b[idx]);
// }
